#include "hip/hip_runtime.h"
#include "tensor_lat_half.h"

int main() {

  intilizeDeviceProp(0);

  if (deviceProp.major < 6) // tesnore unit was added since Volta
    return 1;

  // std::cout << "FP16 operand, FP32 accumalte:\n";
  tensor_lat<half, float>();

  // std::cout << "\nFP16 operand, FP16 accumalte:\n";
  // tensor_lat<half, half>();

  // std::cout << "\n__nv_bfloat16 operand, float accumalte:\n";
  // tensor_lat<__hip_bfloat16, float>();

  //std::cout << "\nint8_t operand, int accumalte:\n";
 // tensor_lat<int8_t,int>();

  return 1;
}
